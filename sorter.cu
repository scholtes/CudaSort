
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define TEST_SIZE 35
#define RAND_RANGE 10
#define BLOCK_WIDTH 4
#define CEILING_DIVIDE(X, Y) (1 + (((X) - 1) / (Y)))

// Computes a blockwise exclusive sum scan
__global__ void partialScan(unsigned int *d_in,
                            unsigned int *d_out,
                            unsigned int *d_total,
                            size_t n)
{
    __shared__ unsigned int temp[BLOCK_WIDTH];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int index = BLOCK_WIDTH * bx + tx;

    if(index < n) {
        temp[tx] = d_in[index];
    }
    __syncthreads();

    // Perform the actual scan
    for(int offset = 1; offset < BLOCK_WIDTH; offset <<= 1) {
        if(tx + offset < BLOCK_WIDTH) {
            temp[tx + offset] += temp[tx];
        }
        __syncthreads();
    }

    // Shift when copying the result so as to make it an exclusive scan
    if(tx + 1 < BLOCK_WIDTH) {
        d_out[index + 1] = temp[tx];
    }
    d_out[0] = 0;

    // Store the total sum of each block
    d_total[bx] = temp[BLOCK_WIDTH - 1];
}

// Compute a map on a partial scan to create a total scan from
__global__ void mapScan(unsigned int *d_array, unsigned int *d_total, size_t n) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int index = BLOCK_WIDTH * bx + tx;

    if(index < n) {
        d_array[index] += d_total[bx];
    }
}


// Compute exclusive sum scan for arbitrary sized array (device pointers as input)
void totalScan(unsigned int *d_in, unsigned int *d_out, size_t n) {
    size_t numBlocks = CEILING_DIVIDE(n, BLOCK_WIDTH);
    unsigned int *d_total;
    hipMalloc(&d_total, sizeof(unsigned int) * numBlocks);

    partialScan<<<numBlocks, BLOCK_WIDTH>>>(d_in, d_out, d_total, n);

    if(numBlocks > 1) {
        unsigned int *d_total_scanned;
        hipMalloc(&d_total_scanned, sizeof(unsigned int) * numBlocks);

        totalScan(d_total, d_total_scanned, numBlocks);
        mapScan<<<numBlocks, BLOCK_WIDTH>>>(d_out, d_total_scanned, n);

        hipFree(d_total_scanned);
    }

    hipFree(d_total);
}


// Do radix sort on d_inputVals and store to d_outputVals. The assosciated
// positions are also moved accordingly
void radix(unsigned int* const d_inputVals,
           unsigned int* const d_inputPos,
           unsigned int* const d_outputVals,
           unsigned int* const d_outputPos,
           const size_t numElems)
{
    unsigned int *inVals;
    unsigned int *inPos;
    unsigned int *zerosPredicate;
    unsigned int *onesPredicate;
    unsigned int *zerosScan;
    unsigned int *onesScan;

    for(int bit = 1; bit <= 1; bit++) {

    }

    hipFree(inVals);
    hipFree(inPos);
    hipFree(zerosPredicate);
    hipFree(onesPredicate);
    hipFree(zerosScan);
    hipFree(onesScan);
}


////////////////////////////////////////////////////////////////////////////////


// Wrapper for totalScan (host pointers as input)
void radixHost(unsigned int* const h_inputVals,
               unsigned int* const h_inputPos,
               unsigned int* const h_outputVals,
               unsigned int* const h_outputPos,
               const size_t numElems)
{
    unsigned int *d_inputVals;
    unsigned int *d_inputPos;
    unsigned int *d_outputVals;
    unsigned int *d_outputPos;
    size_t memsize = sizeof(unsigned int) * numElems;

    hipMalloc(&d_inputVals, memsize);
    hipMalloc(&d_inputPos, memsize);
    hipMalloc(&d_outputVals, memsize);
    hipMalloc(&d_outputPos, memsize);

    hipMemcpy(d_inputVals, h_inputVals, memsize, hipMemcpyHostToDevice);
    hipMemcpy(d_inputPos, h_inputPos, memsize, hipMemcpyHostToDevice);

    radix(d_inputVals, d_inputPos, d_outputVals, d_outputPos, numElems);

    hipMemcpy(h_outputVals, d_outputVals, memsize, hipMemcpyDeviceToHost);
    hipMemcpy(h_outputPos, d_outputPos, memsize, hipMemcpyDeviceToHost);

    hipFree(d_inputVals);
    hipFree(d_inputPos);
    hipFree(d_outputVals);
    hipFree(d_outputPos);
}


int main(int argc, char **argv) {
    unsigned int *h_inVals;
    unsigned int *h_inPos;
    unsigned int *h_outVals;
    unsigned int *h_outPos;

    srand(0);

    size_t memsize = sizeof(unsigned int) * TEST_SIZE;

    h_inVals = (unsigned int*)malloc(memsize);
    h_inPos = (unsigned int*)malloc(memsize);
    h_outVals = (unsigned int*)malloc(memsize);
    h_outPos = (unsigned int*)malloc(memsize);

    // Random test values (seeded)
    for(int i=0; i<TEST_SIZE; i++){ h_inVals[i] = rand() % RAND_RANGE; }
    // Test positions 0 ... TEST_SIZE
    for(int i=0; i<TEST_SIZE; i++){ h_inPos[i] = i; }

    // Compute
    radixHost(h_inVals, h_inPos, h_outVals, h_outPos, TEST_SIZE);

    // Print input
    printf("h_inVals = [ ");
    for(int i=0; i<TEST_SIZE; i++){ printf("%d ", h_inVals[i]); }
    printf("];\nh_inPos = [ ");
    for(int i=0; i<TEST_SIZE; i++){ printf("%d ", h_inPos[i]); }
    printf("];\n");

    // Print output
    printf("h_outVals = [ ");
    for(int i=0; i<TEST_SIZE; i++){ printf("%d ", h_outVals[i]); }
    printf("];\nh_outPos = [ ");
    for(int i=0; i<TEST_SIZE; i++){ printf("%d ", h_outPos[i]); }
    printf("];\n");

    free(h_inVals);
    free(h_inPos);
    free(h_outVals);
    free(h_outPos);
    return 0;
}
